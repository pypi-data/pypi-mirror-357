#include "hip/hip_runtime.h"
/*

    This file is part of MyoQMRI.

    MyoQMRI is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Foobar is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <https://www.gnu.org/licenses/>.
    
    Copyright 2019 Francesco Santini <francesco.santini@unibas.ch>

*/

#ifndef PYCUDA_COMPILE

    #define NECHOES 8
    #define ECHOSPACING 8.5f
    #define T1F 365.0f
    #define T1W 1400.0f
    #define MAGPREP 0

#endif
    
#include <iostream>
#include <math.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/complex.h>
#include <hip/hip_complex.h>

// row-major indexing
#define IDX2C(i,j,nRows) (((j)*(nRows))+(i))
#define IDX2C3(i,j) IDX2C(i,j,3)

using complexType = thrust::complex<float>;
using cuType = hipFloatComplex;

template <typename matrixType, int N>
__device__ void dephase(matrixType *stateMatrix)
{
    for (int i=0; i<N-1; i++)
    {
        stateMatrix[IDX2C3(0,N-i-1)] = stateMatrix[IDX2C3(0,N-i-2)];
        stateMatrix[IDX2C3(1,i)] = stateMatrix[IDX2C3(1,i+1)];
    }
    stateMatrix[IDX2C3(0,0)] = thrust::conj(stateMatrix[IDX2C3(1,0)]);
    stateMatrix[IDX2C3(1, N-1)] = 0.0;
}

template <typename matrixType, int N>
__device__ void relax(matrixType *stateMatrix, const matrixType *relaxMatrix)
{
    matrixType z0 = stateMatrix[IDX2C3(2,0)];
    for (int i=0; i<N*3; i++)
    {
        stateMatrix[i] *= relaxMatrix[i];
    }
    stateMatrix[IDX2C3(2,0)] += 1 - z0;
}

template <typename matrixType, int N>
__device__ void rfMult(const matrixType *tMatrix, const matrixType *stateMatrix, matrixType *outMatrix)
{
    for (auto c=0; c<N; c++)
    {
        for (auto r=0; r<3; r++)
        {
            matrixType sum = 0;
            for (auto rc=0; rc<3; rc++)
            {
                sum += tMatrix[IDX2C3(r,rc)]*stateMatrix[IDX2C3(rc,c)];
            }
            outMatrix[IDX2C3(r,c)] = sum;
        }
    }
}

template <int Nechoes>
__device__ void cpmg(float exc_alpha, float ref_alpha, float T1, float T2, thrust::complex<float> *outVector)
{
    
    const int N = Nechoes;
    const int Nt2 = 2*N;
    const int Nt2p1 = Nt2+1;
    const complexType jp(0.0,1.0);
    const complexType jm(0.0,-1.0);
    
    float alpha_in = M_PI*ref_alpha/180;
    float exc_alpha_in = M_PI*exc_alpha/180;
    
    complexType fa[2];
    
    fa[1] = alpha_in;
    fa[0] = alpha_in;
    
    if (N>1 && MAGPREP)
    {
        fa[0] = (M_PI + fa[1])/2;
    }

    float E1 = exp(-ECHOSPACING/T1/2.0);
    float E2 = exp(-ECHOSPACING/T2/2.0);
    
    complexType RelaxMatrix[3*Nt2p1];
    for (int i=0; i<Nt2p1; i++)
    {
        RelaxMatrix[IDX2C3(0,i)] = complexType(E2,0.0f);
        RelaxMatrix[IDX2C3(1,i)] = complexType(E2,0.0f);
        RelaxMatrix[IDX2C3(2,i)] = complexType(E1,0.0f);
    }
    
    complexType Omega_preRF[3*Nt2p1];
    complexType Omega_postRF[3*Nt2p1];
    
    for (auto i=0; i<3*Nt2p1; i++)
    {
        Omega_postRF[i] = complexType(0.0f, 0.0f);
        Omega_preRF[i] = complexType(0.0f, 0.0f);
    }
    
    Omega_postRF[IDX2C3(0,0)] = sin(exc_alpha_in);
    Omega_postRF[IDX2C3(1,0)] = sin(exc_alpha_in);
    Omega_postRF[IDX2C3(2,0)] = cos(exc_alpha_in);
    
    complexType tMatrix0[3*3];
    tMatrix0[IDX2C3(0,0)] =      pow(cos(fa[0]/2.0),2);
    tMatrix0[IDX2C3(0,1)] =      pow(sin(fa[0]/2.0),2);
    tMatrix0[IDX2C3(0,2)] = jm * sin(fa[0]);
    
    tMatrix0[IDX2C3(1,0)] =      pow(sin(fa[0]/2.0),2);
    tMatrix0[IDX2C3(1,1)] =      pow(cos(fa[0]/2.0),2);
    tMatrix0[IDX2C3(1,2)] = jp * sin(fa[0]);
    
    tMatrix0[IDX2C3(2,0)] = 0.5*jm* sin(fa[0]);
    tMatrix0[IDX2C3(2,1)] = 0.5*jp* sin(fa[0]);
    tMatrix0[IDX2C3(2,2)] =         cos(fa[0]);
    
    complexType tMatrix1[3*3];
    tMatrix1[IDX2C3(0,0)] =      pow(cos(fa[1]/2.0),2);
    tMatrix1[IDX2C3(0,1)] =      pow(sin(fa[1]/2.0),2);
    tMatrix1[IDX2C3(0,2)] = jm * sin(fa[1]);
    
    tMatrix1[IDX2C3(1,0)] =      pow(sin(fa[1]/2.0),2);
    tMatrix1[IDX2C3(1,1)] =      pow(cos(fa[1]/2.0),2);
    tMatrix1[IDX2C3(1,2)] = jp * sin(fa[1]);
    
    tMatrix1[IDX2C3(2,0)] = 0.5*jm* sin(fa[1]);
    tMatrix1[IDX2C3(2,1)] = 0.5*jp* sin(fa[1]);
    tMatrix1[IDX2C3(2,2)] =         cos(fa[1]);
    
    // first relaxation
    //printf("Omega_postRF[0,0]: %f\n", Omega_postRF[0].real());
    relax<complexType, Nt2p1>(Omega_postRF, RelaxMatrix);
    //printf("Omega_postRF[0,0]: %f\n", Omega_postRF[0].real());
    dephase<complexType, Nt2p1>(Omega_postRF);
    //printf("Omega_postRF[0,0]: %f\n", Omega_postRF[0].real());
    
    // first refocusing RF
    
    rfMult<complexType, Nt2p1>(tMatrix0, Omega_postRF, Omega_preRF);
    //printf("Omega_preRF[0,0]: %f\n", Omega_preRF[0].real());
    // relaxation/recovery post refocusing
    relax<complexType, Nt2p1>(Omega_preRF, RelaxMatrix);
    //printf("Omega_preRF[0,0]: %f\n", Omega_preRF[0].real());
    dephase<complexType, Nt2p1>(Omega_preRF);
    //printf("Omega_preRF[0,0]: %f\n", Omega_preRF[0].real());
    
    outVector[0] = thrust::conj(Omega_preRF[IDX2C3(1,0)]);
    //printf("outvector[0] %f\n", outVector[0].real());
    
    thrust::copy(thrust::device, Omega_preRF, Omega_preRF+(3*Nt2p1), Omega_postRF); // copy state to other matrix
    
    for (int pn=1; pn<N; pn++)
    {
        // first relaxation
        
        relax<complexType, Nt2p1>(Omega_postRF, RelaxMatrix);
        dephase<complexType, Nt2p1>(Omega_postRF);
        
        
        // first refocusing RF
        
        rfMult<complexType, Nt2p1>(tMatrix1, Omega_postRF, Omega_preRF);
        
        // relaxation/recovery post refocusing
        relax<complexType, Nt2p1>(Omega_preRF, RelaxMatrix);
        dephase<complexType, Nt2p1>(Omega_preRF);
        
        outVector[pn] = thrust::conj(Omega_preRF[IDX2C3(1,0)]);
        //printf("outvector[%d] %f\n", pn, outVector[pn].real());
        thrust::copy(thrust::device, Omega_preRF, Omega_preRF+(3*Nt2p1), Omega_postRF); // copy state to other matrix
        
    }
    
}

// parameters is nx3: ff, t2, b1
// signals_out is nxNECHOES
extern "C" void __global__ cpmg_sliceprof_B1_FF(unsigned int totalParameters, unsigned int nFlipanglesSP, float T2f, float *flipAnglesEx, float *flipAnglesRef, float *parameters, float *signals_out)
{
    // calculate the cpmg signals for many values of B1 and Fat fractions
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= totalParameters) return;
    float wT2 = parameters[index*3+0];
    float b1 = parameters[index*3+1];
    float ff = parameters[index*3+2];

    //printf("wT2 %f, b1 %f, ff %f\n", wT2, b1, ff);
    
    for (int echo=0; echo<NECHOES; echo++)
    {
        signals_out[index*NECHOES+echo] = 0.0;
    }
    
    for (int nFa = 0; nFa<nFlipanglesSP; nFa++)
    {
        complexType fatSignal[NECHOES];
        complexType waterSignal[NECHOES];
        cpmg<NECHOES>(flipAnglesEx[nFa]*b1, flipAnglesRef[nFa]*b1, T1F, T2f, fatSignal);
        cpmg<NECHOES>(flipAnglesEx[nFa]*b1, flipAnglesRef[nFa]*b1, T1W, wT2, waterSignal);
        //printf("SignalsOut: ");
        for (int echo=0; echo<NECHOES; echo++)
        {
            signals_out[index*NECHOES+echo] += float( (fatSignal[echo]*ff).real() + (waterSignal[echo]*(1-ff)).real() )/nFlipanglesSP;
            //printf("%f ", signals_out[index*NECHOES+echo]);
        }
        //printf("\n");
    }
}

// Test

#ifndef PYCUDA_COMPILE

#define T2F 151.0f

#define NFF 1
#define NT2 6
#define NB1 2

#define minT2 20.0
#define maxT2 80.0

#define minB1 0.6
#define maxB1 1.4

#define minFF 0.0
#define maxFF 1.0

__global__ void createParams(float *params, float *spExc, float *spRef)
{
    spExc[0] = 45;
    spExc[1] = 90;
    spExc[2] = 45;
    
    spRef[0] = 90;
    spRef[1] = 180;
    spRef[2] = 90;
    
    int paramIndex = 0;
    // initialize x and y arrays on the host
    for (int nFF = 0; nFF<NFF; nFF++)
    {
        for (int nT2 = 0; nT2 < NT2; nT2++)
        {
            for (int nB1 = 0; nB1 < NB1; nB1++)
            {
                //printf("paramIndex %d\n", paramIndex);
                params[paramIndex++] = float(nT2)*(maxT2-minT2)/NT2 + minT2;
                params[paramIndex++] = float(nB1)*(maxB1-minB1)/NB1 + minB1;
                params[paramIndex++] = float(nFF)*(maxFF-minFF)/NFF + minFF;
            }
        }
    }
}

int main(void)
{
    

  unsigned int Nparams = NT2*NFF*NB1;
  
  float *params, *signals;

  float *spExc;
  float *spRef;
  
  // Allocate Unified Memory accessible from CPU or GPU
  hipMalloc((void**)&params, 3*Nparams*sizeof(float));
  
  hipMalloc((void**)&spExc, 3*sizeof(float));
  hipMalloc((void**)&spRef, 3*sizeof(float));
 
  std::cout << "Creating param space" << std::endl << std::flush;
  
  createParams<<<1,1>>>(params, spExc, spRef);
  
  hipDeviceSynchronize();
  
  float *h_params = (float*)malloc(Nparams*3*sizeof(float));
  hipMemcpy(h_params, params, Nparams*3*sizeof(float), hipMemcpyDeviceToHost);
  
  std::cout << h_params[0] << ", " << h_params[1] << ", " << h_params[2] << std::endl;
  
  hipMalloc((void**)&signals, NECHOES*Nparams*sizeof(float));
  
  std::cout << "Creating signals" << std::endl << std::flush;
  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int nBlocks = ceil( float(Nparams)/blockSize );
  //cpmg_sliceprof_B1_FF<<< nBlocks, blockSize >>>(Nparams,  3, T2F, spExc, spRef, params, signals);
  cpmg_sliceprof_B1_FF<<< 1, 1 >>>(Nparams,  3, T2F, spExc, spRef, params, signals);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  float *h_signals = (float*)malloc(Nparams*NECHOES*sizeof(float));
  hipMemcpy(h_signals, signals, Nparams*NECHOES*sizeof(float), hipMemcpyDeviceToHost);
  
  std::cout << "Signals created" << std::endl << std::flush;

  std::cout << "Example: ";
  
  for (int i=0; i<NECHOES; i++)
  {
      std::cout << h_signals[0*NECHOES + i] << ", ";
  }
  std::cout << std::endl;
  
  hipFree(params);
  hipFree(signals);
  hipFree(spExc);
  hipFree(spRef);
  
  return 0;
}

#endif
